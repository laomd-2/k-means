#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <cfloat>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <fstream>
#include "reduce.cuh"
#include "common.cuh"
#include "io.cuh"

__device__ inline void sum_in_block(float* local, float* global, int k)
{
    for (int i = 0; i < k; i++)
        local[i] = blockReduce(local[i], thrust::plus<float>(), 0.0f);
    if (threadIdx.x == 0) {
        for (int i = 0; i < k; i++) {
            int offset = i * gridDim.x;
            // k * num_blocks
            // 这里虽然不能合并访存，但在compute_new_means归并时可以合并访存
            global[offset + blockIdx.x] = local[i];
        }
    }
}

__global__ void assign_clusters(const float* data_x, const float* data_y, int* label, int data_size,
                                const float* means_x, const float* means_y, 
                                float* new_sums_x, float* new_sums_y, float* counts, int k)
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    const int width = blockDim.x * gridDim.x;

    // 假设k不大于16
    float local_sum_x[16] = {0.0f};
    float local_sum_y[16] = {0.0f};
    float local_count[16] = {0.0f};

    for (int index = id; index < data_size; index += width) {
        const float x = data_x[index];
        const float y = data_y[index];
        int best_cluster = find_nearest_cluster(x, y, means_x, means_y, k);
        label[index] = best_cluster;
        local_sum_x[best_cluster] += x;
        local_sum_y[best_cluster] += y;
        local_count[best_cluster]++;
    }

    sum_in_block(local_sum_x, new_sums_x, k);
    sum_in_block(local_sum_y, new_sums_y, k);
    sum_in_block(local_count, counts, k);
}

__device__ inline float sum_in_device1block(const float* in, int width) {
    float s = 0.0f;
    int wid = threadIdx.x / warpSize;
    int laneid = threadIdx.x & (warpSize - 1);
    // 一个warp算一行
    int offset = wid * width;
    for (int i = laneid; i < width; i += warpSize) {
        s += in[offset + i];
    }
    s = warpReduce(s, thrust::plus<float>());
    return s;
}

// Each thread is one cluster, which just recomputes its coordinates as the mean
// of all points assigned to it.
__global__ void compute_new_means(float* new_means_x, float* new_means_y,
                       float* new_sum_x, float* new_sum_y, float* counts, 
                       int blocks, float* max_diff)
{
    int cluster = threadIdx.x / warpSize;
    int laneid = threadIdx.x & (warpSize - 1);

    float sum_x = sum_in_device1block(new_sum_x, blocks);
    float sum_y = sum_in_device1block(new_sum_y, blocks);
    float count = max(1.0f, sum_in_device1block(counts, blocks));

    __shared__ float max_val[32];
    if (laneid == 0) {
        float mean_x = new_means_x[cluster], mean_y= new_means_y[cluster];
        float new_mean_x = sum_x / count, new_mean_y = sum_y / count;
        new_means_x[cluster] = new_mean_x;
        new_means_y[cluster] = new_mean_y;
        max_val[cluster] = squared_l2_distance(mean_x, mean_y, new_mean_x, new_mean_y);
    }
    if (cluster == 0) {
        // 归约最大值
        float val = warpReduce(max_val[laneid], thrust::maximum<float>());
        if (laneid == 0)    *max_diff = val;
    }
}

int main(int argc, const char *argv[])
{
    thrust::host_vector<float> h_x;
    thrust::host_vector<float> h_y;
    int k;
    k = read_data(argv[1], h_x);
    k = read_data(argv[2], h_y);
    int number_of_elements = h_x.size();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    thrust::device_vector<float> d_x(h_x);
    thrust::device_vector<float> d_y(h_y);
    thrust::device_vector<float> d_mean_x(k);
    thrust::device_vector<float> d_mean_y(k);
    thrust::device_vector<float> distance(1 + number_of_elements);

    srand(time(NULL));
    int index = rand() % number_of_elements;
    d_mean_x[0] = d_x[index];
    d_mean_y[0] = d_y[index];

    const int threads = 1024;
    int blocks = std::min(64, (number_of_elements + threads - 1) / threads);
    for (int i = 1; i < k; i++) {
        get_distance<<<blocks, threads>>>(
            thrust::raw_pointer_cast(d_x.data()),
            thrust::raw_pointer_cast(d_y.data()),
            thrust::raw_pointer_cast(distance.data()),
            number_of_elements, 
            thrust::raw_pointer_cast(d_mean_x.data()),
            thrust::raw_pointer_cast(d_mean_y.data()),
            i
        );
        hipDeviceSynchronize();
        thrust::inclusive_scan(distance.begin(), distance.end(), distance.begin());
        float seed = (rand() % number_of_elements) / (float)number_of_elements;
        choice_cluster<<<blocks, threads>>>(
            thrust::raw_pointer_cast(d_x.data()),
            thrust::raw_pointer_cast(d_y.data()),
            thrust::raw_pointer_cast(distance.data()),
            number_of_elements, 
            thrust::raw_pointer_cast(d_mean_x.data()),
            thrust::raw_pointer_cast(d_mean_y.data()),
            i, seed
        );
        hipDeviceSynchronize();
    }
    
    thrust::device_vector<float> d_sums_x(k * blocks, 0.0f), d_sums_y(k * blocks, 0.0f), 
                                 d_counts(k * blocks, 0.0f);
    thrust::device_vector<int> d_label(number_of_elements, 0);
    float *d_s;
    hipMalloc(&d_s, sizeof(float));

    float tol = 1e-4f, s = tol + 1.0f;
    int number_of_iterations = 300, iteration;
    for (iteration = 0; s > tol && iteration < number_of_iterations; ++iteration)
    {
        assign_clusters<<<blocks, threads, k * sizeof(float)>>>(
            thrust::raw_pointer_cast(d_x.data()),
            thrust::raw_pointer_cast(d_y.data()),
            thrust::raw_pointer_cast(d_label.data()),
            number_of_elements,
            thrust::raw_pointer_cast(d_mean_x.data()),
            thrust::raw_pointer_cast(d_mean_y.data()),
            thrust::raw_pointer_cast(d_sums_x.data()),
            thrust::raw_pointer_cast(d_sums_y.data()),
            thrust::raw_pointer_cast(d_counts.data()),
            k
        );
        hipDeviceSynchronize();

        compute_new_means<<<1, k * 32>>>(
            thrust::raw_pointer_cast(d_mean_x.data()),
            thrust::raw_pointer_cast(d_mean_y.data()),
            thrust::raw_pointer_cast(d_sums_x.data()),
            thrust::raw_pointer_cast(d_sums_y.data()),
            thrust::raw_pointer_cast(d_counts.data()),
            blocks, d_s
        );
        hipDeviceSynchronize();
        hipMemcpy(&s, d_s, sizeof(float), hipMemcpyDeviceToHost);
    }
    hipFree(d_s);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << milliseconds / 1000.0 << "s with " << iteration << " rounds" << std::endl;

    output(argv[3], d_label, d_mean_x, d_mean_y);
}